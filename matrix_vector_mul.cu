
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void matrix_vector_product(float *a, float *v, float *res, int matrix_size) {
    float sum = 0.0;
    for (int j = 0; j < blockDim.x; j++) {
        sum += a[threadIdx.x * blockDim.x + j] * v[j];
    } 
    res[threadIdx.x] = sum;
}

int main(int argc, char **argv) {
    float *a, *v, *res; 
    float *a_gpu, *v_gpu, *res_gpu;
    int matrix_size = atoi(argv[1]);
    
    a = (float *) malloc(matrix_size * matrix_size * sizeof(float));
    v = (float *) malloc(matrix_size * sizeof(float));
    res = (float *) malloc(matrix_size * sizeof(float));
    
    for (int i = 0; i < matrix_size; i++) {
        for (int j = 0; j < matrix_size; j++) {
            a[i * matrix_size + j] = (float) i * matrix_size + j;
        }
        v[i] = (float) i;
    }

    hipMalloc((void **) &a_gpu, matrix_size * matrix_size * sizeof(float));
    hipMalloc((void **) &v_gpu, matrix_size * sizeof(float));
    hipMalloc((void **) &res_gpu, matrix_size * sizeof(float));

    hipMemcpy(a_gpu, a, matrix_size * matrix_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(v_gpu, v, matrix_size * sizeof(float), hipMemcpyHostToDevice);

    clock_t st, en;
    st = clock();
    matrix_vector_product<<<1, matrix_size>>>(a_gpu, v_gpu, res_gpu, matrix_size);
    en = clock();

    hipMemcpy(res, res_gpu, matrix_size * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < matrix_size; i++) {
        printf("%.2f\n", res[i]);
    }
    
    printf("Time taken by GPU: %f seconds\n", (double)(en - st) / CLOCKS_PER_SEC);

    free(a);
    free(v);
    free(res);
    hipFree(a_gpu);
    hipFree(v_gpu);
    hipFree(res_gpu);

    return 0;
}