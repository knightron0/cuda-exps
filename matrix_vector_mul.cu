
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void matrix_vector_product(float *a, float *v, float *res, int matrix_size) {
    int index = (blockDim.x * blockIdx.x) + threadIdx.x;
    // index = threadIdx.x;
    if (index < matrix_size) {
        float sum = 0.0;
        for (int j = 0; j < matrix_size; j++) {
            sum += a[index * matrix_size + j] * v[j];
        } 
        res[index] = sum;
        // printf("%d\n", index);
    }
}

int main(int argc, char **argv) {
    float *a, *v, *res; 
    float *a_gpu, *v_gpu, *res_gpu;
    int matrix_size = atoi(argv[1]);
    
    a = (float *) malloc(matrix_size * matrix_size * sizeof(float));
    v = (float *) malloc(matrix_size * sizeof(float));
    res = (float *) malloc(matrix_size * sizeof(float));
    
    for (int i = 0; i < matrix_size; i++) {
        for (int j = 0; j < matrix_size; j++) {
            a[i * matrix_size + j] = (float) i * matrix_size + j;
        }
        v[i] = (float) i;
    }

    hipMalloc((void **) &a_gpu, matrix_size * matrix_size * sizeof(float));
    hipMalloc((void **) &v_gpu, matrix_size * sizeof(float));
    hipMalloc((void **) &res_gpu, matrix_size * sizeof(float));

    hipMemcpy(a_gpu, a, matrix_size * matrix_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(v_gpu, v, matrix_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 grid_size(10);
    dim3 block_size((matrix_size / 10) + 1);
    // printf("grid size: %d %d\n", grid_size.x, grid_size.y);
    // printf("block size: %d %d\n", block_size.x, block_size.y);
    clock_t st, en;
    st = clock();
    matrix_vector_product<<<grid_size, block_size>>>(a_gpu, v_gpu, res_gpu, matrix_size);
    en = clock();

    hipMemcpy(res, res_gpu, matrix_size * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < matrix_size; i++) {
        printf("%.2f\n", res[i]);
    }
    
    printf("Time taken by GPU: %f seconds\n", (double)(en - st) / CLOCKS_PER_SEC);

    free(a);
    free(v);
    free(res);
    hipFree(a_gpu);
    hipFree(v_gpu);
    hipFree(res_gpu);

    return 0;
}