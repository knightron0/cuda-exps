
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_cuda() {
    printf("Hello from CUDA\n");
    printf("Block Index X: %d, Block Index Y: %d, Thread Index X: %d, Thread Index Y: %d\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
}

int main() {
    hello_cuda<<<2, 2>>>();
    hipDeviceSynchronize();

    return 0;
}